#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <chrono>
#include <thread>
#include <memory>
#include <thrust/for_each.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/detail/config.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/random.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/transform.h>
#include <thrust/transform_reduce.h>
#include <thrust/extrema.h>
#include <thrust/execution_policy.h>
#include "include/thread_pool.h"

struct Add2
{
    template <typename Tuple>
    __host__ __device__
    void operator()(Tuple t)
    {
        // C[i] = A[i] + B[i]
        thrust::get<2>(t) = thrust::get<0>(t) + thrust::get<1>(t);
    }
};

struct Subtract2
{
    template <typename Tuple>
    __host__ __device__
    void operator()(Tuple t)
    {
        // C[i] = A[i] - B[i]
        thrust::get<2>(t) = thrust::get<0>(t) - thrust::get<1>(t);
    }
};

struct Multiply2
{
    template <typename Tuple>
    __host__ __device__
    void operator()(Tuple t)
    {
        // C[i] = A[i] * B[i]
        thrust::get<2>(t) = thrust::get<0>(t) * thrust::get<1>(t);
    }
};

struct Divide2
{
    template <typename Tuple>
    __host__ __device__
    void operator()(Tuple t)
    {
        // C[i] = A[i] / B[i]
        thrust::get<2>(t) = thrust::get<0>(t) / thrust::get<1>(t);
    }
};

template <typename T>
struct MinMaxPair
{
   T min_val;
   T max_val;
};

template <typename T>
struct MinMaxUnaryOp
  : public thrust::unary_function< T, MinMaxPair<T> >
{
   __host__ __device__
   MinMaxPair<T> operator()(const T& x) const
   {
       MinMaxPair<T> result;
       result.min_val = x;
       result.max_val = x;
       return result;
   }
};

template <typename T>
struct MinMaxBinaryOp
  : public thrust::binary_function< MinMaxPair<T>, MinMaxPair<T>, MinMaxPair<T> >
{
    __host__ __device__
    MinMaxPair<T> operator()(const MinMaxPair<T>& x, const MinMaxPair<T>& y) const
    {
        MinMaxPair<T> result;
        result.min_val = thrust::min(x.min_val, y.min_val);
        result.max_val = thrust::max(x.max_val, y.max_val);
        return result;
    }
};

thrust::host_vector<float> getRandomVector(const size_t N,
                                           unsigned int seed = thrust::default_random_engine::default_seed)
{
    
    thrust::minstd_rand rng(seed);
    thrust::random::normal_distribution<float> dist(128.0f, 32.0f);
    thrust::host_vector<float> temp(N);
    for(size_t i = 0; i < N; i++)
    {
        temp[i] = dist(rng);
    }
    return temp;
}

template <typename T>
struct Square
{
    __host__ __device__
    T operator()(const T& x) const 
    {
        return x * x;
    }
};

/**
 * \brief Main processing pipeline to mimic a forward pass
 */
void processingPipeline(int w, int h)
{
    int N = w * h;
    thrust::device_vector<float> A = getRandomVector(N, 10);
    thrust::device_vector<float> B = getRandomVector(N, 71);
    thrust::device_vector<float> C = getRandomVector(N, 24);
    thrust::device_vector<float> D = getRandomVector(N, 63);

    thrust::device_vector<float> AB(N);
    thrust::device_vector<float> CD(N);

    thrust::device_vector<float> AplusB(N);
    thrust::device_vector<float> CplusD(N);

    thrust::device_vector<float> F(N);
    thrust::device_vector<float> G(N);

    thrust::device_vector<float> H(N);

    MinMaxUnaryOp<float>  unaryOp;
    MinMaxBinaryOp<float> binaryOp;
    Square<float>         squareOp;
    thrust::plus<float>   plusOp;

    // A * B
    thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(A.begin(), B.begin(), AB.begin())),
                     thrust::make_zip_iterator(thrust::make_tuple(A.end(), B.end(), AB.end())),
                     Multiply2());

    // C * D
    thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(C.begin(), D.begin(), CD.begin())),
                     thrust::make_zip_iterator(thrust::make_tuple(C.end(), D.end(), CD.end())),
                     Multiply2());

    // A + B
    thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(A.begin(), B.begin(), AplusB.begin())),
                     thrust::make_zip_iterator(thrust::make_tuple(A.end(), B.end(), AplusB.end())),
                     Add2());

    // C + D
    thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(C.begin(), D.begin(), CplusD.begin())),
                     thrust::make_zip_iterator(thrust::make_tuple(C.end(), D.end(), CplusD.end())),
                     Add2());

    // 2 * (A + B)
    thrust::transform(AplusB.begin(), AplusB.end(),
                      thrust::constant_iterator<float>(2.f),
                      AplusB.begin(),
                      thrust::multiplies<float>());
    
    // 2 * (C + D)
    thrust::transform(CplusD.begin(), CplusD.end(),
                      thrust::constant_iterator<float>(2.f),
                      CplusD.begin(),
                      thrust::multiplies<float>());

    // F = (A * B) / (2 * (A + B)
    thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(AB.begin(), AplusB.begin(), F.begin())),
                     thrust::make_zip_iterator(thrust::make_tuple(AB.end(), AplusB.end(), F.end())),
                     Divide2());

    // G = (C * D) / (2 * (C + D)
    thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(CD.begin(), CplusD.begin(), G.begin())),
                     thrust::make_zip_iterator(thrust::make_tuple(CD.end(), CplusD.end(), G.end())),
                     Divide2());

    // MinMax(F)
    MinMaxPair<float> minmaxF = thrust::transform_reduce(F.begin(), F.end(), unaryOp, unaryOp(F[0]), binaryOp);

    // MinMax(G)
    MinMaxPair<float> minmaxG = thrust::transform_reduce(G.begin(), G.end(), unaryOp, unaryOp(G[0]), binaryOp);

    // H = F - G
    thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(F.begin(), G.begin(), H.begin())),
                     thrust::make_zip_iterator(thrust::make_tuple(F.end(), G.end(), H.end())),
                     Subtract2());

    // Norm(H)
    float norm = std::sqrt(thrust::transform_reduce(H.begin(), H.end(), squareOp, 0, plusOp));
}


/**
 * @brief Driver
 */
int main(int argc, char *argv[])
{
    int numRuns = 1;
    if (argc == 2)
    {
        numRuns = std::stoi(argv[1]);
    }

    ThreadPool threadPool(8);
    auto start = std::chrono::steady_clock::now();

    if (numRuns > 1)
    {
        threadPool.Start();
        for (int i = 0; i < numRuns; ++i)
        {
            threadPool.QueueWorkItem(i + 1, []() { processingPipeline(640, 384); });
        }

        while(threadPool.QSize() > 0)
        {
            std::this_thread::sleep_for(std::chrono::milliseconds(3));
        }

        threadPool.RequestStop();
        threadPool.Wait();
        threadPool.Clear();
    }
    else
    {
        processingPipeline(640, 384);
    }

    std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
    auto processingTime = static_cast<long long>(std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count());
    std::cout << "Total Processing Time: " << processingTime << " ms." << std::endl;
    std::cout << "Time Per Thread: " << processingTime / numRuns << " ms." << std::endl;

    return 0;
}